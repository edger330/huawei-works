#include "hip/hip_runtime.h"
/*Copyright (c) 2012 The Broad Institute

*Permission is hereby granted, free of charge, to any person
*obtaining a copy of this software and associated documentation
*files (the "Software"), to deal in the Software without
*restriction, including without limitation the rights to use,
*copy, modify, merge, publish, distribute, sublicense, and/or sell
*copies of the Software, and to permit persons to whom the
*Software is furnished to do so, subject to the following
*conditions:

*The above copyright notice and this permission notice shall be
*included in all copies or substantial portions of the Software.

*THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
*EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
*OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
*NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
*HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
*WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
*FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR
*THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/


#include "common_data_structure.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <omp.h>

using namespace std;

template<class NUMBER>
__device__ NUMBER set_match_to_match_prob(int i, int d, NUMBER * d_jacobianLogTable, NUMBER * d_matchToMatchProb)
{
    NUMBER output;
    NUMBER result;
    int minQual = d;
    int maxQual = i;
    if (i <= d)
    {
        minQual = i;
        maxQual = d;
    }
    NUMBER small = ((NUMBER)-0.1)*minQual;
    NUMBER big =  ((NUMBER)-0.1)*maxQual;
    NUMBER diff = big - small;
    if (isinf(small) == -1 || isinf(big) == -1 || diff >= ((NUMBER)(MAX_JACOBIAN_TOLERANCE))){
        result = big;
    } else {
        NUMBER d = (NUMBER)(diff * ((NUMBER)JACOBIAN_LOG_TABLE_INV_STEP));
        int ind = (d > ((NUMBER)0.0)) ? (int) (d + ((NUMBER)0.5)) : (int) (d - ((NUMBER)0.5));
        result = big + d_jacobianLogTable[ind];
    }
    output = (MAX_QUAL < maxQual) ? ((NUMBER)1.0) - pow(((NUMBER)10), result) : d_matchToMatchProb[((maxQual * (maxQual + 1)) >> 1) + minQual];
    return output;
}

__device__ int find_index(int ROWS, int COLS, int position_x, int position_y)
{
    if (position_x + position_y < ROWS) {
        return (position_x + position_y) * (position_x + position_y + 1) / 2 + position_y;
    } else if (ROWS - 1 - position_x + COLS - 1 - position_y < ROWS) {
        return ROWS * COLS - 1 - (ROWS + COLS - 2 -position_x - position_y) * (ROWS + COLS - 1 - position_x - position_y) / 2 - (COLS - 1 - position_y);
    } else {
        return ROWS * (position_y - 1) - (ROWS - position_x - 1) * (ROWS - 1) + ROWS * (ROWS + 1) / 2;;
    }
}

template <class NUMBER>
__global__ void compute(char * d_tc_rs_all,
                        char * d_tc_hap_all,
                        char * d_tc_q_all,
                        char * d_tc_i_all,
                        char * d_tc_d_all,
                        char * d_tc_c_all,
                        NUMBER * d_ph2pr,
                        NUMBER * d_matchToMatchProb,
                        NUMBER * d_jacobianLogTable,
                        int * d_ROWS, int * d_COLS,
                        int * d_row_offset, int * d_col_offset,
                        NUMBER * d_result,
                        NUMBER INITIAL_CONSTANT)
{
    int index = blockIdx.x;
    int ROWS = d_ROWS[index];
    int COLS = d_COLS[index];
    char * d_tc_rs = d_tc_rs_all + d_row_offset[index];
    char * d_tc_hap = d_tc_hap_all + d_col_offset[index];
    char * d_tc_q = d_tc_q_all + d_row_offset[index];
    char * d_tc_i = d_tc_i_all + d_row_offset[index];
    char * d_tc_d = d_tc_d_all + d_row_offset[index];
    char * d_tc_c = d_tc_c_all + d_row_offset[index];
    int shift = threadIdx.x;

    extern __shared__ __align__(sizeof(NUMBER)) unsigned char temp[];
    NUMBER * common_buffer = reinterpret_cast<NUMBER *>(temp);

    // initialize
    NUMBER * M = common_buffer;
    NUMBER * X = common_buffer + ROWS * COLS;
    NUMBER * Y = X + ROWS * COLS;
    NUMBER * p = Y + ROWS * COLS;

    p[MM] = NUMBER(0.0);
    p[GapM] = NUMBER(0.0);
    p[MX] = NUMBER(0.0);
    p[XX] = NUMBER(0.0);
    p[MY] = NUMBER(0.0);
    p[YY] = NUMBER(0.0);


    if (shift < ROWS) {
        // init p
        int _i = d_tc_i[shift-1] & 127;
        int _d = d_tc_d[shift-1] & 127;
        int _c = d_tc_c[shift-1] & 127;
        p[shift * 6 + MM] = set_match_to_match_prob(_i, _d, d_jacobianLogTable, d_matchToMatchProb);
        p[shift * 6 + GapM] = NUMBER(1.0) - d_ph2pr[_c];
        p[shift * 6 + MX] = d_ph2pr[_i];
        p[shift * 6 + XX] = d_ph2pr[_c];
        p[shift * 6 + MY] = d_ph2pr[_d];
        p[shift * 6 + YY] = d_ph2pr[_c];

        // init row of MXY
        int index = find_index(ROWS, COLS, shift, 0);
        int last_index = find_index(ROWS, COLS, shift - 1, 0);
        M[index] = NUMBER(0.0);
        X[index] = X[last_index] * p[shift * 6 + XX];
        Y[index] = NUMBER(0.0);
    }

    if (shift > COLS) {
        // init col of MXY
        int index = find_index(ROWS, COLS, 0, shift);
        M[index] = NUMBER(0.0);
        X[index] = NUMBER(0.0);
        Y[index] = INITIAL_CONSTANT / (COLS - 1);
    }

    __syncthreads();

    if (shift < ROWS - 1) {
        int pos_x = 1;
        int pos_y = 1;
        int num_thread = 0;
        for (int i = 0; i < ROWS + COLS - 3; i++) {
            if (i < ROWS - 1) // first triangle
            {
                num_thread ++;
                if (i != 0) {
                    pos_x ++;
                }
            } else if (i < COLS - 1) // second area
            {
                pos_y ++;
            } else // third triangle
            {
                num_thread --;
                pos_y ++;
            }
            if (shift < num_thread) {
                int r = pos_x - shift;
                int c = pos_y + shift;
                char _rs = d_tc_rs[r - 1];
                char _hap = d_tc_hap[c - 1];
                int _q = d_tc_q[r - 1] & 127;
                NUMBER distm = d_ph2pr[_q];
                if (_rs == _hap || _rs == 'N' || _hap == 'N')
                    distm = NUMBER(1.0) - distm;
                else
                    distm = distm / 3;

                int position = find_index(ROWS, COLS, r, c);
                int left = find_index(ROWS, COLS, r , c - 1);
                int up = find_index(ROWS, COLS, r - 1, c);
                int leftup = find_index(ROWS, COLS, r - 1, c - 1);

                M[position] = distm * (M[leftup] * p[r * 6 + MM] + X[leftup] * p[r * 6 + GapM] + Y[leftup] * p[r * 6 + GapM]);

                X[position] = M[up] * p[r * 6 + MX] + X[up] * p[r * 6 + XX];

                Y[position] = M[left] * p[r * 6 + MY] + Y[left] * p[r * 6 + YY];
            }
            __syncthreads();
        }
    }

    if (shift == 0) {
        NUMBER result = NUMBER(0.0);
        for (int c = 0; c < COLS; c++) {
            int pos = find_index(ROWS, COLS, ROWS - 1, c);
            result += M[pos] + X[pos];
        }

        d_result[index] = result;
    }
}

template<class NUMBER>
__global__ void initialize(char * d_tc_i_all,
                           char * d_tc_d_all,
                           char * d_tc_c_all,
                           NUMBER * d_M,
                           NUMBER * d_X,
                           NUMBER * d_Y,
                           NUMBER * d_p,
                           NUMBER * d_ph2pr,
                           NUMBER * d_matchToMatchProb,
                           NUMBER * d_jacobianLogTable,
                           int * d_ROWS, int * d_COLS,
                           int * d_row_offset, int * d_col_offset, int * d_MXY_offset, int * d_p_offset,
                           NUMBER INITIAL_CONSTANT)
{
    int index = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int ROWS = d_ROWS[index];
    int COLS = d_COLS[index];
    char * d_tc_i = d_tc_i_all + d_row_offset[index];
    char * d_tc_d = d_tc_d_all + d_row_offset[index];
    char * d_tc_c = d_tc_c_all + d_row_offset[index];

    NUMBER * M = d_M + d_MXY_offset[index];
    NUMBER * X = d_X + d_MXY_offset[index];
    NUMBER * Y = d_Y + d_MXY_offset[index];
    NUMBER * p = d_p + d_p_offset[index];
    p[MM] = NUMBER(0.0);
    p[GapM] = NUMBER(0.0);
    p[MX] = NUMBER(0.0);
    p[XX] = NUMBER(0.0);
    p[MY] = NUMBER(0.0);
    p[YY] = NUMBER(0.0);

    for (int r = 1; r < ROWS; r++)
    {
        int _i = d_tc_i[r-1] & 127;
        int _d = d_tc_d[r-1] & 127;
        int _c = d_tc_c[r-1] & 127;
        p[r * 6 + MM] = set_match_to_match_prob(_i, _d, d_jacobianLogTable, d_matchToMatchProb);
        p[r * 6 + GapM] = NUMBER(1.0) - d_ph2pr[_c];
        p[r * 6 + MX] = d_ph2pr[_i];
        p[r * 6 + XX] = d_ph2pr[_c];
        p[r * 6 + MY] = d_ph2pr[_d];
        p[r * 6 + YY] = d_ph2pr[_c];
    }

    for (int c = 0; c < COLS; c++)
    {
        int index = find_index(ROWS, COLS, 0, c);
        M[index] = NUMBER(0.0);
        X[index] = NUMBER(0.0);
        Y[index] = INITIAL_CONSTANT / (COLS - 1);
    }

    for (int r = 1; r < ROWS; r++)
    {
        int index = find_index(ROWS, COLS, r, 0);
        int last_index = find_index(ROWS, COLS, r - 1, 0);
        M[index] = NUMBER(0.0);
        X[index] = X[last_index] * p[r * 6 + XX];
        Y[index] = NUMBER(0.0);
    }
}

template<class NUMBER>
__global__ void cuda_compute_batch_prob(char * d_tc_rs_all,
                                        char * d_tc_hap_all,
                                        char * d_tc_q_all,
                                        NUMBER * d_M,
                                        NUMBER * d_X,
                                        NUMBER * d_Y,
                                        NUMBER * d_p,
                                        NUMBER * d_ph2pr,
                                        int * d_ROWS, int * d_COLS,
                                        int * d_row_offset, int * d_col_offset, int * d_MXY_offset, int * d_p_offset)
{
    int index = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int ROWS = d_ROWS[index];
    int COLS = d_COLS[index];
    char * d_tc_rs = d_tc_rs_all + d_row_offset[index];
    char * d_tc_hap = d_tc_hap_all + d_col_offset[index];
    char * d_tc_q = d_tc_q_all + d_row_offset[index];


    NUMBER * M = d_M + d_MXY_offset[index];
    NUMBER * X = d_X + d_MXY_offset[index];
    NUMBER * Y = d_Y + d_MXY_offset[index];
    NUMBER * p = d_p + d_p_offset[index];

    for (int r = 1; r < ROWS; r++)
        for (int c = 1; c < COLS; c++)
        {
            char _rs = d_tc_rs[r-1];
            char _hap = d_tc_hap[c-1];
            int _q = d_tc_q[r-1] & 127;
            NUMBER distm = d_ph2pr[_q];
            if (_rs == _hap || _rs == 'N' || _hap == 'N')
                distm = NUMBER(1.0) - distm;
            else
                distm = distm/3;

            int position = find_index(ROWS, COLS, r, c);
            int left = find_index(ROWS, COLS, r , c - 1);
            int up = find_index(ROWS, COLS, r - 1, c);
            int leftup = find_index(ROWS, COLS, r - 1, c - 1);

            M[position] = distm * (M[leftup] * p[r * 6 + MM] + X[leftup] * p[r * 6 + GapM] + Y[leftup] * p[r * 6 + GapM]);

            X[position] = M[up] * p[r * 6 + MX] + X[up] * p[r * 6 + XX];

            Y[position] = M[left] * p[r * 6 + MY] + Y[left] * p[r * 6 + YY];

//            M[r * COLS + c] = distm * (M[(r-1) * COLS + (c-1)] * p[r * 6 + MM] + X[(r-1) * COLS + (c-1)] * p[r * 6 + GapM] + Y[(r-1) * COLS + (c-1)] * p[r * 6 + GapM]);
//
//            X[r * COLS + c] = M[(r-1) * COLS + c] * p[r * 6 + MX] + X[(r-1) * COLS + c] * p[r * 6 + XX];
//
//            Y[r * COLS + c] = M[r * COLS + (c-1)] * p[r * 6 + MY] + Y[r * COLS + (c-1)] * p[r * 6 + YY];
        }
}

template<class NUMBER>
__global__ void diagonal_compute(char * d_tc_rs_all,
                                 char * d_tc_hap_all,
                                 char * d_tc_q_all,
                                 NUMBER * d_M,
                                 NUMBER * d_X,
                                 NUMBER * d_Y,
                                 NUMBER * d_p,
                                 NUMBER * d_ph2pr,
                                 int * d_ROWS, int * d_COLS,
                                 int * d_row_offset, int * d_col_offset,
                                 int * d_MXY_offset, int * d_p_offset)
{
    int index = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int shift = threadIdx.x + threadIdx.y * blockDim.x;
    int ROWS = d_ROWS[index];
    int COLS = d_COLS[index];
    char * d_tc_rs = d_tc_rs_all + d_row_offset[index];
    char * d_tc_hap = d_tc_hap_all + d_col_offset[index];
    char * d_tc_q = d_tc_q_all + d_row_offset[index];

    NUMBER * M = d_M + d_MXY_offset[index];
    NUMBER * X = d_X + d_MXY_offset[index];
    NUMBER * Y = d_Y + d_MXY_offset[index];
    NUMBER * p = d_p + d_p_offset[index];

    extern __shared__ __align__(sizeof(NUMBER)) unsigned char temp[];
    NUMBER * shared_p = reinterpret_cast<NUMBER *>(temp);
    for (int z = threadIdx.x; z < ROWS * 6; z += 32){
        shared_p[z] = p[z];
    }
    __syncthreads();

    if (shift < ROWS - 1) {
        int pos_x = 1;
        int pos_y = 1;
        int num_thread = 0;
        for (int i = 0; i < ROWS + COLS - 3; i++) {
            if (i < ROWS - 1) // first triangle
            {
                num_thread ++;
                if (i != 0) {
                    pos_x ++;
                }
            } else if (i < COLS - 1) // second area
            {
                pos_y ++;
            } else // third triangle
            {
                num_thread --;
                pos_y ++;
            }
            if (shift < num_thread) {
                int r = pos_x - shift;
                int c = pos_y + shift;
                // computing
                char _rs = d_tc_rs[r - 1];
                char _hap = d_tc_hap[c - 1];
                int _q = d_tc_q[r - 1] & 127;
                NUMBER distm = d_ph2pr[_q];
                if (_rs == _hap || _rs == 'N' || _hap == 'N')
                    distm = NUMBER(1.0) - distm;
                else
                    distm = distm / 3;

                int position = find_index(ROWS, COLS, r, c);
                int left = find_index(ROWS, COLS, r , c - 1);
                int up = find_index(ROWS, COLS, r - 1, c);
                int leftup = find_index(ROWS, COLS, r - 1, c - 1);

//                NUMBER tempM = distm * (M[leftup] * p[r * 6 + MM] + X[leftup] * p[r * 6 + GapM] + Y[leftup] * p[r * 6 + GapM]);
                M[position] = distm * (M[leftup] * shared_p[r * 6 + MM] + X[leftup] * shared_p[r * 6 + GapM] + Y[leftup] * shared_p[r * 6 + GapM]);

//                NUMBER tempX= M[up] * p[r * 6 + MX] + X[up] * p[r * 6 + XX];
                X[position] = M[up] * shared_p[r * 6 + MX] + X[up] * shared_p[r * 6 + XX];

//                NUMBER tempY = M[left] * p[r * 6 + MY] + Y[left] * p[r * 6 + YY];
                Y[position] = M[left] * shared_p[r * 6 + MY] + Y[left] * shared_p[r * 6 + YY];
            }
            __syncthreads();
        }
    }
}

template<class NUMBER>
__global__ void compute_result(NUMBER * d_M,
                               NUMBER * d_X,
                               int * d_ROWS, int * d_COLS,
                               int * d_MXY_offset,
                               NUMBER * d_result)
{
    int index = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int ROWS = d_ROWS[index];
    int COLS = d_COLS[index];

    NUMBER * M = d_M + d_MXY_offset[index];
    NUMBER * X = d_X + d_MXY_offset[index];

    NUMBER result = NUMBER(0.0);
    for (int c = 0; c < COLS; c++)
    {
        int pos = find_index(ROWS, COLS, ROWS - 1, c);
        result += M[pos] + X[pos];
    }

    d_result[index] = result;
}

resultSet * cuda_compute_full_prob(dataSet * data) {

//    std::cout << "1" << std::endl;
//    std::cout << "size : " << data->numHaps * data->numReads << std::endl;
    Context<float > ctx;
    resultSet * ret = new resultSet();
    ret->taskID = data->taskID;
    ret->sampleID = data->sampleID;
    ret->numHaps = data->numHaps;
    ret->numReads = data->numReads;
    ret->result = (float *)malloc(ret->numHaps * ret->numReads * sizeof(float));
    // rs q i d c
    int sizeRead = 4096 / ret->numHaps;
    int size = sizeRead * ret->numHaps;
//    std::cout << "size : " << size << std::endl;
//    std::cout << "size read : " << sizeRead << std::endl;

//    std::cout << "2" << std::endl;

    int readsDataLens = 0;
    int hapsDataLens = 0;
    int * readsOffset = (int *)malloc(ret->numReads * sizeof(int));
    int * hapsOffset = (int *)malloc(ret->numHaps * sizeof(int));
    for (int i = 0; i < ret->numReads; i++) {
        readsOffset[i] = readsDataLens;
        readsDataLens += data->readsLens[i];
//        std::cout << "readsLens[] : " << data->readsLens[i] << std::endl;
    }

    for (int i = 0; i < ret->numHaps; i++) {
        hapsOffset[i] = hapsDataLens;
        hapsDataLens += data->hapsLens[i];
//        std::cout << "hapsLens[] : " << data->hapsLens[i] << std::endl;
    }

    int result_offset = 0;
//    std::cout << "3" << std::endl;
    for (int batch_index = 0 ; batch_index < ret->numReads; batch_index += sizeRead) {
        int Vec_size;
        if (batch_index + sizeRead < ret->numReads) {
//            std::cout << "enter 1" << std::endl;
            Vec_size = size;
        } else {
//            std::cout << "enter 2" << std::endl;
            Vec_size = (ret->numReads - batch_index) * ret->numHaps;
        }
        int readSize = Vec_size / ret->numHaps;
//        std::cout << "readSize : " << readSize << std::endl;

        int *d_ROWS;
        int *d_COLS;
        int *ROWS_all = (int *) malloc(Vec_size * sizeof(int));
        int *COLS_all = (int *) malloc(Vec_size * sizeof(int));
        int *d_row_offset;
        int *d_col_offset;
        int *row_offset = (int *) malloc(Vec_size * sizeof(int));
        int *col_offset = (int *) malloc(Vec_size * sizeof(int));
        int *d_MXY_offset;
        int *d_p_offset;
        int *MXY_offset = (int *) malloc(Vec_size * sizeof(int));
        int *p_offset = (int *) malloc(Vec_size * sizeof(int));
        int max_thread = 0;
        int max_cols = 0;

        int R_off = 0;
        int C_off = 0;
        int mxy_off = 0;
        int p_off = 0;
        for (int read_index = 0; read_index < readSize; read_index++) {
            for (int hap_index = 0; hap_index < ret->numHaps; hap_index++) {
                int ROWS = data->readsLens[read_index + batch_index] + 1;
                int COLS = data->hapsLens[hap_index] + 1;
                row_offset[read_index * ret->numHaps + hap_index] = R_off;
                col_offset[read_index * ret->numHaps + hap_index] = C_off;
                MXY_offset[read_index * ret->numHaps + hap_index] = mxy_off;
                p_offset[read_index * ret->numHaps + hap_index] = p_off;
                R_off += (ROWS - 1);
                C_off += (COLS - 1);
                mxy_off += (ROWS * COLS);
                p_off += (ROWS * 6);
                ROWS_all[read_index * ret->numHaps + hap_index] = ROWS;
                COLS_all[read_index * ret->numHaps + hap_index] = COLS;
                if (max_thread < ROWS - 1) {
                    max_thread = ROWS - 1;
                }
                if (max_cols < COLS) {
                    max_cols = COLS;
                }
            }
        }
//        std::cout << "4" << std::endl;
        char *d_tc_rs_all;
        char *d_tc_hap_all;
        char *d_tc_q_all;
        char *d_tc_i_all;
        char *d_tc_d_all;
        char *d_tc_c_all;
        float * d_M;
        float * d_X;
        float * d_Y;
        float * d_p;
        char *tc_rs_all = (char *)malloc(R_off * sizeof(char));
        char *tc_hap_all = (char *)malloc(C_off * sizeof(char));
        char *tc_q_all = (char *)malloc(R_off * sizeof(char));
        char *tc_i_all = (char *)malloc(R_off * sizeof(char));
        char *tc_d_all = (char *)malloc(R_off * sizeof(char));
        char *tc_c_all = (char *)malloc(R_off * sizeof(char));
//        std::cout << "R_OFF : " << R_off << std::endl;
//        std::cout << "C_OFF : " << C_off << std::endl;
//        std::cout << "row_offset[]" << row_offset[(readSize - 1) * ret->numHaps + ret->numHaps -1] << std::endl;
//        std::cout << "col_offset[]" << col_offset[(readSize - 1) * ret->numHaps + ret->numHaps -1] << std::endl;
//        std::cout << "enter memcpy" << std::endl;
        for (int read_index = 0; read_index < readSize; read_index++) {
            for (int hap_index = 0; hap_index < ret->numHaps; hap_index++) {
//                std::cout << "ROWS : " << ROWS_all[read_index * ret->numHaps + hap_index] - 1 << std::endl;
                memcpy(tc_rs_all + row_offset[read_index * ret->numHaps + hap_index], data->readsData + readsOffset[read_index + batch_index], (ROWS_all[read_index * ret->numHaps + hap_index] - 1) * sizeof(char));
//                std::cout << "row_offset : " << row_offset[read_index * ret->numHaps + hap_index] << std::endl;
                memcpy(tc_hap_all + col_offset[read_index * ret->numHaps + hap_index], data->hapsData + hapsOffset[hap_index], (COLS_all[read_index * ret->numHaps + hap_index] - 1) * sizeof(char));
//                std::cout << "read_offset : " << readsOffset[read_index + batch_index] << std::endl;
                memcpy(tc_q_all + row_offset[read_index * ret->numHaps + hap_index], data->readsData + readsOffset[read_index + batch_index] + readsDataLens, (ROWS_all[read_index * ret->numHaps + hap_index] - 1) * sizeof(char));
//                std::cout << "readsDataLen : " << readsDataLens << std::endl;
                memcpy(tc_i_all + row_offset[read_index * ret->numHaps + hap_index], data->readsData + readsOffset[read_index + batch_index] + 2 * readsDataLens, (ROWS_all[read_index * ret->numHaps + hap_index] - 1) * sizeof(char));
                memcpy(tc_d_all + row_offset[read_index * ret->numHaps + hap_index], data->readsData + readsOffset[read_index + batch_index] + 3 * readsDataLens, (ROWS_all[read_index * ret->numHaps + hap_index] - 1) * sizeof(char));
                memcpy(tc_c_all + row_offset[read_index * ret->numHaps + hap_index], data->readsData + readsOffset[read_index + batch_index] + 4 * readsDataLens, (ROWS_all[read_index * ret->numHaps + hap_index] - 1) * sizeof(char));
            }
        }
//        std::cout << "5" << std::endl;
        hipMalloc((void **) &d_M, mxy_off * sizeof(float));
        hipMalloc((void **) &d_X, mxy_off * sizeof(float));
        hipMalloc((void **) &d_Y, mxy_off * sizeof(float));
        hipMalloc((void **) &d_p, p_off * sizeof(float));
        hipMalloc((void **) &d_tc_rs_all, R_off * sizeof(char));
        hipMalloc((void **) &d_tc_hap_all, C_off * sizeof(char));
        hipMalloc((void **) &d_tc_q_all, R_off * sizeof(char));
        hipMalloc((void **) &d_tc_i_all, R_off * sizeof(char));
        hipMalloc((void **) &d_tc_d_all, R_off * sizeof(char));
        hipMalloc((void **) &d_tc_c_all, R_off * sizeof(char));
        hipMemcpy(d_tc_rs_all, tc_rs_all, R_off * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_tc_hap_all, tc_hap_all, C_off * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_tc_q_all, tc_q_all, R_off * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_tc_i_all, tc_i_all, R_off * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_tc_d_all, tc_d_all, R_off * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_tc_c_all, tc_c_all, R_off * sizeof(char), hipMemcpyHostToDevice);

        hipMalloc((void **) &d_ROWS, Vec_size * sizeof(int));
        hipMalloc((void **) &d_COLS, Vec_size * sizeof(int));
        hipMemcpy(d_ROWS, ROWS_all, Vec_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_COLS, COLS_all, Vec_size * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc((void **) &d_row_offset, Vec_size * sizeof(int));
        hipMalloc((void **) &d_col_offset, Vec_size * sizeof(int));
        hipMemcpy(d_row_offset, row_offset, Vec_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_col_offset, col_offset, Vec_size * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc((void **) &d_MXY_offset, Vec_size * sizeof(int));
        hipMalloc((void **) &d_p_offset, Vec_size * sizeof(int));
        hipMemcpy(d_MXY_offset, MXY_offset, Vec_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_p_offset, p_offset, Vec_size * sizeof(int), hipMemcpyHostToDevice);

        float * d_ph2pr;
        float * d_matchToMatchProb;

        float * d_jacobianLogTable;
        float * d_result;
        float * result = (float *)malloc(Vec_size * sizeof(float));

        hipMalloc((void **) &d_ph2pr, 128 * sizeof(float));
        hipMalloc((void **) &d_matchToMatchProb, (((MAX_QUAL + 1) * (MAX_QUAL + 2)) >> 1) * sizeof(float));
        hipMalloc((void **) &d_jacobianLogTable, JACOBIAN_LOG_TABLE_SIZE * sizeof(float));
        hipMalloc((void **) &d_result, Vec_size * sizeof(float));

        hipMemcpy(d_ph2pr, ctx.ph2pr, 128 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_matchToMatchProb, ctx.matchToMatchProb, (((MAX_QUAL + 1) * (MAX_QUAL + 2)) >> 1) * sizeof(float),
                   hipMemcpyHostToDevice);
        hipMemcpy(d_jacobianLogTable, ctx.jacobianLogTable, JACOBIAN_LOG_TABLE_SIZE * sizeof(float),
                   hipMemcpyHostToDevice);
//        std::cout << "6" << std::endl;
        initialize << < Vec_size, 1 >> >
                                  (d_tc_i_all, d_tc_d_all, d_tc_c_all, d_M, d_X, d_Y, d_p, d_ph2pr, d_matchToMatchProb, d_jacobianLogTable, d_ROWS, d_COLS, d_row_offset, d_col_offset, d_MXY_offset, d_p_offset, ctx.INITIAL_CONSTANT);
        hipDeviceSynchronize();

        int warps = max_thread / 32;
        if (warps * 32 < max_thread) {
            warps++;
        }
        dim3 block(32, warps);
        diagonal_compute << < Vec_size, block, (max_thread + 1) * 6 * sizeof(float) >> >
                                               (d_tc_rs_all, d_tc_hap_all, d_tc_q_all, d_M, d_X, d_Y, d_p, d_ph2pr, d_ROWS, d_COLS, d_row_offset, d_col_offset, d_MXY_offset, d_p_offset);
        hipDeviceSynchronize();

        compute_result << < Vec_size, 1 >> > (d_M, d_X, d_ROWS, d_COLS, d_MXY_offset, d_result);
        hipDeviceSynchronize();

        hipMemcpy(result, d_result, Vec_size * sizeof(float), hipMemcpyDeviceToHost);
//        std::cout << "7" << std::endl;
        hipFree(d_tc_rs_all);
        hipFree(d_tc_hap_all);
        hipFree(d_tc_q_all);
        hipFree(d_tc_i_all);
        hipFree(d_tc_d_all);
        hipFree(d_tc_c_all);
//        std::cout << "finish device qidc" << std::endl;
        hipFree(d_M);
        hipFree(d_X);
        hipFree(d_Y);
        hipFree(d_p);
//        std::cout << "finish MXYp" << std::endl;
        free(tc_rs_all);
        free(tc_hap_all);
        free(tc_q_all);
        free(tc_i_all);
        free(tc_d_all);
        free(tc_c_all);
//        std::cout << "finish host qidc" << std::endl;
        hipFree(d_ph2pr);
        hipFree(d_matchToMatchProb);
        hipFree(d_jacobianLogTable);
        hipFree(d_result);
        hipFree(d_ROWS);
        hipFree(d_COLS);
//        std::cout << "finish device ph2COLS" << std::endl;
        free(ROWS_all);
        free(COLS_all);
//        std::cout << "finish host ROW COL" << std::endl;
        hipFree(d_row_offset);
        hipFree(d_col_offset);
//        std::cout << "finish device offset" << std::endl;
        free(row_offset);
        free(col_offset);
//        std::cout << "finish host offset" << std::endl;
        hipFree(d_MXY_offset);
        hipFree(d_p_offset);
//        std::cout << "finish deivce MXYp offset" << std::endl;
        free(MXY_offset);
        free(p_offset);
//        std::cout << "finish host MXYp offset" << std::endl;
//        std::cout << "result_offset : " << result_offset << std::endl;
        memcpy(ret->result + result_offset, result, Vec_size * sizeof(float));
        free(result);
        result_offset += Vec_size;
//        std::cout << "8" << std::endl;
    }
//    std::cout << "9" << std::endl;
    free(readsOffset);
    free(hapsOffset);
    return ret;
}

//template double * cuda_compute_full_prob<double>(vector<testcase>& tcVec);
//template float * cuda_compute_full_prob<float>(vector<testcase>& tcVec);

